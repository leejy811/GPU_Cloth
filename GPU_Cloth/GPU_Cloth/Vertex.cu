#include "Vertex.h"

Vertex::Vertex()
{

}

Vertex::~Vertex()
{

}

void Vertex::InitDeviceMem(uint numVertex)
{
	hipMalloc(&d_restPos, sizeof(REAL3) * numVertex);	hipMemset(d_restPos, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_Pos, sizeof(REAL3) * numVertex);		hipMemset(d_Pos, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_Pos1, sizeof(REAL3) * numVertex);		hipMemset(d_Pos1, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_Vel, sizeof(REAL3) * numVertex);		hipMemset(d_Vel, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_vNormal, sizeof(REAL3) * numVertex);	hipMemset(d_vNormal, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_InvMass, sizeof(REAL) * numVertex);		hipMemset(d_InvMass, 0, sizeof(REAL) * numVertex);
	hipMalloc(&d_SatMass, sizeof(REAL) * numVertex);		hipMemset(d_SatMass, 0, sizeof(REAL) * numVertex);
	hipMalloc(&d_vSaturation, sizeof(REAL) * numVertex);		hipMemset(d_vSaturation, 0, sizeof(REAL) * numVertex);
	hipMalloc(&d_Adhesion, sizeof(REAL3) * numVertex);		hipMemset(d_Adhesion, 0, sizeof(REAL3) * numVertex);
	hipMalloc(&d_vAngle, sizeof(REAL) * numVertex);		hipMemset(d_vAngle, 0, sizeof(REAL) * numVertex);
}

void Vertex::copyToDevice(const Mesh& mesh)
{
	hipMemcpy(d_restPos, &mesh.h_pos[0], sizeof(REAL3) * mesh.h_pos.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_Pos, &mesh.h_pos[0], sizeof(REAL3) * mesh.h_pos.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_Pos1, &mesh.h_pos1[0], sizeof(REAL3) * mesh.h_pos1.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_Vel, &mesh.h_vel[0], sizeof(REAL3) * mesh.h_vel.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_vNormal, &mesh.h_vNormal[0], sizeof(REAL3) * mesh.h_vNormal.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_InvMass, &mesh.h_invMass[0], sizeof(REAL) * mesh.h_invMass.size(), hipMemcpyHostToDevice);

	d_nbFaces.copyFromHost(mesh.h_nbVFaces);
	d_nbVertices.copyFromHost(mesh.h_nbVertices);
}

void Vertex::copyToHost(Mesh& mesh)
{
	hipMemcpy(&mesh.h_pos[0], d_Pos, sizeof(REAL3) * mesh.h_pos.size(), hipMemcpyDeviceToHost);
	hipMemcpy(&mesh.h_vNormal[0], d_vNormal, sizeof(REAL3) * mesh.h_vNormal.size(), hipMemcpyDeviceToHost);
	hipMemcpy(&mesh.h_vAngle[0], d_vAngle, sizeof(REAL) * mesh.h_vAngle.size(), hipMemcpyDeviceToHost);
}

void Vertex::FreeDeviceMem(void)
{
	hipFree(d_restPos);
	hipFree(d_Pos);
	hipFree(d_Pos1);
	hipFree(d_Vel);
	hipFree(d_vNormal);
	hipFree(d_InvMass);
	hipFree(d_SatMass);
	hipFree(d_vSaturation);
	hipFree(d_Adhesion);
	hipFree(d_vAngle);

	d_nbFaces.clear();
	d_nbVertices.clear();
}
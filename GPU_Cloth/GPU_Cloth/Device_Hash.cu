#include "Device_Hash.cuh"

Device_Hash::Device_Hash()
{

}

Device_Hash::~Device_Hash()
{

}

void Device_Hash::InitParam(ClothParam clothParam)
{
	_param._gridRes = clothParam._gridRes;
	_param._numParticle = clothParam._numVertices;

	hipMemcpyToSymbol(HIP_SYMBOL(hashParam), &_param, sizeof(HashParam));
}

void Device_Hash::SetHashTable_kernel(REAL3* pos)
{
	CalculateHash_Kernel(pos);
	SortParticle_Kernel();
	FindCellStart_Kernel();
}

void Device_Hash::CalculateHash_Kernel(REAL3* pos)
{
	uint numThreads, numBlocks;
	ComputeGridSize(_param._numParticle, 256, numBlocks, numThreads);

	CalculateHash_D << <numBlocks, numThreads >> >
		(d_gridHash, d_gridIdx, pos);
}

void Device_Hash::SortParticle_Kernel()
{
	thrust::sort_by_key(thrust::device_ptr<uint>(d_gridHash),
		thrust::device_ptr<uint>(d_gridHash + _param._numParticle),
		thrust::device_ptr<uint>(d_gridIdx));
}

void Device_Hash::FindCellStart_Kernel()
{
	uint numThreads, numBlocks;
	ComputeGridSize(_param._numParticle, 256, numBlocks, numThreads);

	uint smemSize = sizeof(uint) * (numThreads + 1);
	FindCellStart_D << <numBlocks, numThreads, smemSize >> >
		(d_gridHash, d_gridIdx, d_cellStart, d_cellEnd);
}

void Device_Hash::InitDeviceMem(uint numParticle)
{
	uint numCell = _param._gridRes * _param._gridRes * _param._gridRes;

	hipMalloc(&d_gridHash, sizeof(uint) * numParticle);	hipMemset(d_gridHash, 0, sizeof(uint) * numParticle);
	hipMalloc(&d_gridIdx, sizeof(uint) * numParticle);	hipMemset(d_gridIdx, 0, sizeof(uint) * numParticle);
	hipMalloc(&d_cellStart, sizeof(uint) * numCell);	hipMemset(d_cellStart, 0, sizeof(uint) * numCell);
	hipMalloc(&d_cellEnd, sizeof(uint) * numCell);	hipMemset(d_cellEnd, 0, sizeof(uint) * numCell);
}

void Device_Hash::FreeDeviceMem(void)
{
	hipFree(d_gridHash);
	hipFree(d_gridIdx);
	hipFree(d_cellStart);
	hipFree(d_cellEnd);
}
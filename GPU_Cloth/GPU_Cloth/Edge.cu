#include "hip/hip_runtime.h"
#include "Edge.h"

Edge::Edge()
{

}

Edge::~Edge()
{

}

void Edge::InitDeviceMem(uint numEdge)
{
	hipMalloc(&d_edgeIdx, sizeof(uint2) * numEdge);	hipMemset(d_edgeIdx, 0, sizeof(uint2) * numEdge);
	hipMalloc(&d_restAngle, sizeof(REAL) * numEdge);	hipMemset(d_restAngle, 0, sizeof(REAL) * numEdge);
	hipMalloc(&d_cotWeight, sizeof(REAL) * numEdge);	hipMemset(d_cotWeight, 0, sizeof(REAL) * numEdge);
}

void Edge::copyToDevice(const Mesh& mesh)
{
	hipMemcpy(d_edgeIdx, &mesh.h_edgeIdx[0], sizeof(uint2) * mesh.h_edgeIdx.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_restAngle, &mesh.h_restAngle[0], sizeof(REAL) * mesh.h_restAngle.size(), hipMemcpyHostToDevice);
	d_nbEFaces.copyFromHost(mesh.h_nbEFaces);
}

void Edge::copyToHost(Mesh& mesh)
{
	hipMemcpy(&mesh.h_cotWeight[0], d_cotWeight, sizeof(REAL) * mesh.h_cotWeight.size(), hipMemcpyDeviceToHost);
}

void Edge::FreeDeviceMem(void)
{
	hipFree(d_edgeIdx);
	hipFree(d_restAngle);
	d_nbEFaces.clear();
}
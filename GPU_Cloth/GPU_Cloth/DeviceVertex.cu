#include "hip/hip_runtime.h"
#include "DeviceVertex.cuh"

DeviceVertex::DeviceVertex()
{

}

DeviceVertex::~DeviceVertex()
{

}

void DeviceVertex::InitMem()
{
	hipMalloc(&_dPos, sizeof(float3) * _numVertices);			hipMemset(_dPos, 0, sizeof(float3) * _numVertices);
	hipMalloc(&_dPos1, sizeof(float3) * _numVertices);		hipMemset(_dPos1, 0, sizeof(float3) * _numVertices);
	hipMalloc(&_dVel, sizeof(float3) * _numVertices);			hipMemset(_dVel, 0, sizeof(float3) * _numVertices);
	hipMalloc(&_dNormal, sizeof(float3) * _numVertices);		hipMemset(_dNormal, 0, sizeof(float3) * _numVertices);
	hipMalloc(&_dInvMass, sizeof(float) * _numVertices);		hipMemset(_dInvMass, 0, sizeof(float) * _numVertices);
}

void DeviceVertex::FreeMem()
{
	hipFree(_dPos);
	hipFree(_dPos1);
	hipFree(_dVel);
	hipFree(_dNormal);
	hipFree(_dInvMass);
}

void DeviceVertex::CopyToDevice()
{
	hipMemcpy(_dPos, &_pos[0], sizeof(float3) * _numVertices, hipMemcpyHostToDevice);
	hipMemcpy(_dPos1, &_pos1[0], sizeof(float3) * _numVertices, hipMemcpyHostToDevice);
	hipMemcpy(_dVel, &_vel[0], sizeof(float3) * _numVertices, hipMemcpyHostToDevice);
	hipMemcpy(_dNormal, &_normal[0], sizeof(float3) * _numVertices, hipMemcpyHostToDevice);
	hipMemcpy(_dInvMass, &_invMass[0], sizeof(float) * _numVertices, hipMemcpyHostToDevice);
}

void DeviceVertex::CopyToHost()
{
	hipMemcpy(&_pos[0], _dPos, sizeof(float3) * _numVertices, hipMemcpyDeviceToHost);
	hipMemcpy(&_pos1[0], _dPos1, sizeof(float3) * _numVertices, hipMemcpyDeviceToHost);
	hipMemcpy(&_vel[0], _dVel, sizeof(float3) * _numVertices, hipMemcpyDeviceToHost);
	hipMemcpy(&_normal[0], _dNormal, sizeof(float3) * _numVertices, hipMemcpyDeviceToHost);
	hipMemcpy(&_invMass[0], _dInvMass, sizeof(float) * _numVertices, hipMemcpyDeviceToHost);
}
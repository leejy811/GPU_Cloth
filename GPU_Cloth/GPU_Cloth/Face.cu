#include "hip/hip_runtime.h"
#include "Face.h"

Face::Face()
{

}

Face::~Face()
{

}

void Face::InitDeviceMem(uint numFace)
{
	hipMalloc(&d_faceIdx, sizeof(uint3) * numFace);	hipMemset(d_faceIdx, 0, sizeof(uint3) * numFace);
	hipMalloc(&d_faceAABB, sizeof(AABB) * numFace);	hipMemset(d_faceAABB, 0, sizeof(AABB) * numFace);
	hipMalloc(&d_fNormal, sizeof(REAL3) * numFace);	hipMemset(d_fNormal, 0, sizeof(REAL3) * numFace);
}

void Face::copyToDevice(const Mesh& mesh)
{
	hipMemcpy(d_faceIdx, &mesh.h_faceIdx[0], sizeof(uint3) * mesh.h_faceIdx.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_fNormal, &mesh.h_fNormal[0], sizeof(REAL3) * mesh.h_fNormal.size(), hipMemcpyHostToDevice);
}

void Face::FreeDeviceMem(void)
{
	hipFree(d_faceIdx);
	hipFree(d_faceAABB);
	hipFree(d_fNormal);
}
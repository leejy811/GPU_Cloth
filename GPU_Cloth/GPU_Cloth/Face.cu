#include "hip/hip_runtime.h"
#include "Face.h"

Face::Face()
{

}

Face::~Face()
{

}

void Face::InitDeviceMem(uint numFace)
{
	hipMalloc(&d_faceIdx, sizeof(uint3) * numFace);	hipMemset(d_faceIdx, 0, sizeof(uint3) * numFace);
	hipMalloc(&d_faceAABB, sizeof(AABB) * numFace);	hipMemset(d_faceAABB, 0, sizeof(AABB) * numFace);
	hipMalloc(&d_fNormal, sizeof(REAL3) * numFace);	hipMemset(d_fNormal, 0, sizeof(REAL3) * numFace);
	hipMalloc(&d_fSaturation, sizeof(REAL) * numFace);	hipMemset(d_fSaturation, 0, sizeof(REAL) * numFace);
	hipMalloc(&d_fDripbuf, sizeof(REAL) * numFace);	hipMemset(d_fDripbuf, 0, sizeof(REAL) * numFace);
	hipMalloc(&d_fDripThres, sizeof(REAL) * numFace);	hipMemset(d_fDripThres, 0, sizeof(REAL) * numFace);
}

void Face::copyToDevice(const Mesh& mesh)
{
	hipMemcpy(d_faceIdx, &mesh.h_faceIdx[0], sizeof(uint3) * mesh.h_faceIdx.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_fNormal, &mesh.h_fNormal[0], sizeof(REAL3) * mesh.h_fNormal.size(), hipMemcpyHostToDevice);

	d_nbFace.copyFromHost(mesh.h_nbFFaces);
}

void Face::copyToHost(Mesh& mesh)
{
	hipMemcpy(&mesh.h_fSaturation[0], d_fSaturation, sizeof(REAL) * mesh.h_fSaturation.size(), hipMemcpyDeviceToHost);
}

void Face::FreeDeviceMem(void)
{
	hipFree(d_faceIdx);
	hipFree(d_faceAABB);
	hipFree(d_fNormal);
	hipFree(d_fSaturation);
	hipFree(d_fDripbuf);
	hipFree(d_fDripThres);
}